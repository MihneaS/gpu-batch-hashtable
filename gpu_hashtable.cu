#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

#define MAX_LOADFACTOR 0.75f

__device__
int my_hash(int data, int limit) {
    return ((long)abs(data) * 184014863) % 203676871 % limit;
}


__device__
void add_entry(entry *data, int data_size, int key, int value)
{
    int hashed_key;
    hashed_key = my_hash(key, data_size);
    int old;

    for (int i = 0; i < data_size; i++) {
        old = atomicCAS(&(data[hashed_key].key), 0, key);
        if (old == 0 || old == key) {
            break;
        }
        hashed_key = (hashed_key + 1) % data_size;
    }
    if (old == 0 || old == key) {
        data[hashed_key].value = value;
    }
}

__device__
int get_entry(entry *data, int data_size, int key)
{
    int hashed_key;
    hashed_key = my_hash(key, data_size);
    int key_detected;

    for (int i = 0; i < data_size; i++) {
        key_detected = data[hashed_key].key;
        if (key_detected == key) {
            break;
        }
        hashed_key = (hashed_key + 1) % data_size;
    }
    if (key_detected == data[hashed_key].key) {
        return data[hashed_key].value;
    }
    return -1;
}

__global__
void GPUinsert(int *keys, int *values, int keys_size, entry *data, int data_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < keys_size; i += stride) {
        add_entry(data, data_size, keys[i], values[i]);
    }
}

__global__
void GPUget(int *keys, int *return_values, int keys_size, entry *data, int data_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < keys_size; i += stride) {
        return_values[i] = get_entry(data, data_size, keys[i]);
    }
}

__global__
void GPUreinsert(entry *dst, entry *src, int dst_size, int src_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < src_size; i += stride) {
        if (src[i].key != 0) {
            add_entry(dst, dst_size, src[i].key, src[i].value);
        }
    }
}


/* INIT HASH
 */

GpuHashTable::GpuHashTable(int size) {
    this->size = size;
    int cmr =hipMalloc((void**) &(this->data), size * sizeof(entry));
    hipMemset(data, 0, size * sizeof(entry));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
    hipFree(data);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
    entry* bigger_data;
    hipMalloc((void**) &bigger_data, numBucketsReshape * sizeof(entry));
    hipMemset(bigger_data, 0, numBucketsReshape * sizeof(entry));

    //mut totul in bigger_data
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    hipDeviceSynchronize(); // make sure previous stuff finished
    GPUreinsert<<<numBlocks, blockSize>>>(bigger_data, data, numBucketsReshape, size);
    hipDeviceSynchronize(); // wait to finish the process


    hipFree(data);
    data = bigger_data;
    size = numBucketsReshape;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {
    //manage hashtable's size
    int new_size = size;
    int current_occupied = occupied();
    cerr << "size:" << size << endl;
    cerr << "current_occupied:" << current_occupied << endl;
    cerr << "numKeys:" << numKeys << endl;
    cerr << "loadFactor:" << (current_occupied + numKeys) * 1.0f / new_size << endl;
    while ((current_occupied + numKeys) * 1.0f / new_size > MAX_LOADFACTOR) {
        new_size *= 2;
    }
    cerr << "new_size:" << new_size << endl;
    if (new_size != size) {
        reshape(new_size);
    }

    // alloc gpu memory
    int *gpu_keys;
    int *gpu_values;
    hipMalloc(&gpu_keys, numKeys * sizeof(int));
    hipMalloc(&gpu_values, numKeys * sizeof(int));

    // transfer from input from cpu to gpu
    hipMemcpy(gpu_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_values, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

    // run insertion
    int blockSize = 256;
    int numBlocks = (numKeys + blockSize - 1) / blockSize;
    GPUinsert<<<numBlocks, blockSize>>>(gpu_keys, gpu_values, numKeys, data, size);

    return true; // ce ar trebui sa intoarca aceasta functie?
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
    // alloc gpu memory
    int *gpu_keys;
    int *gpu_values;
    hipMalloc(&gpu_keys, numKeys * sizeof(int));
    hipMalloc(&gpu_values, numKeys * sizeof(int));

    // transfer input from cpu to gpu
    hipMemcpy(gpu_keys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

    // run extraction
    int blockSize = 256;
    int numBlocks = (numKeys + blockSize - 1) / blockSize;
    hipDeviceSynchronize(); // make sure previous stuff finished
    GPUget<<<numBlocks, blockSize>>>(gpu_keys, gpu_values, numKeys, data, size);

    // retrive result
    int *results = (int*)malloc(numKeys * sizeof(int));
    hipDeviceSynchronize(); // wait  for GPU to process
    hipMemcpy(results, gpu_values, numKeys * sizeof(int), hipMemcpyDeviceToHost);

    return results;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */

__global__
void kernel_occupied(entry *data, int size, int *result) {
    int occupied = 0;
    for (int i = 0; i < size; i++) {
        if (data[i].key != 0) {
            occupied++;
        }
    }
    *result = occupied;
}

int GpuHashTable::occupied() {
    int *result;
    int to_return;
    hipMalloc(&result, sizeof(int));
    hipDeviceSynchronize(); // make sure previous stuff finished
    kernel_occupied<<<1, 1>>>(data, size, result);
    hipDeviceSynchronize(); // wait  for GPU to process
    hipMemcpy(&to_return, result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(result);
    return to_return;
}


float GpuHashTable::loadFactor() {
    float tmp = 1.0f * occupied() / size; // no larger than 1.0f = 100%
    return tmp;
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
